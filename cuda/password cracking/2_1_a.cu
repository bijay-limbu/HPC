#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/****************************************************************************
 * An experiment with cuda kernel invocation parameters. 2x3x4 threads on  
 * one block should yield 24 kernel invocations.
 *
 * Compile with:
 *   nvcc -o 2_1_a 2_1_a.cu
 *
 * Dr Kevan Buckley, University of Wolverhampton, January 2018
 *****************************************************************************/
__device__ int is_a_match(char *attempt){
char password1[] ="AA1111";
char password2[] ="BB2222";
char password3[] ="CC3333";
char password4[] ="DD4444";

char *a = attempt;
char *b = attempt;
char *c = attempt;
char *d = attempt;
char *pass1 = password1;
char *pass2 = password2;
char *pass3 = password3;
char *pass4 = password4;

 while(*a ==*pass1){
  if(*a == '\0')
 {
 printf("password:%s\n", password1);
 break;
 }
  a++;
  pass1++;
  }
 while(*b ==*pass2){
  if(*b == '\0')
 {
 printf("password:%s\n", password2);
 break;
  }
 b++;
 pass2++;
 }
 while(*c ==*pass3){
  if(*c == '\0')
{
printf("password:%s\n", password3);
break;
}
c++;
pass3++;
}
while(*d ==*pass4){
  if(*d == '\0')
{
printf("password: %s\n",password4);
return 1;
}
d++;
pass4++;
}
return 0;
}

__global__ void kernel(){
char i1, i2, i3, i4;

char password[7];
password[6] ='\0';

int i = blockIdx.x +65;
int j = threadIdx.x+65;
char firstMatch =i;
char secondMatch =j;

password[0] =firstMatch;
password[1] =secondMatch;
for(i1='0'; i1<='9'; i1++){
for(i2='0'; i2<='9'; i2++){
for(i3='0'; i3<='9'; i3++){
for(i4='0'; i4<='9'; i4++){
password[2] =i1;
password[3] =i2;
password[4] =i3;
password[5] =i4;
if(is_a_match(password)){
}
else{
//printf("tried: %s\n",password);
}
     }
    }
  }
}
}
int time_difference(struct timespec *start, struct timespec *finish,long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

 struct timespec start, finish;  
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel<<<26,26>>>();
hipDeviceSynchronize();
 

 clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
         (time_elapsed/1.0e9));
return 0;
}
