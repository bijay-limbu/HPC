#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>

/****************************************************************************
 * An experiment with cuda kernel invocation parameters. 2x3x4 threads on  
 * one block should yield 24 kernel invocations.
 *
 * Compile with:
 *   nvcc -o 2_1_a_2d 2_1_a_2digit.cu
 *
 * Dr Kevan Buckley, University of Wolverhampton, January 2018
 *****************************************************************************/
__device__ int is_a_match(char *attempt){
char password1[] ="AA11";
char password2[] ="BB22";
char password3[] ="CC33";
char password4[] ="DD44";

char *a = attempt;
char *b = attempt;
char *c = attempt;
char *d = attempt;
char *pass1 = password1;
char *pass2 = password2;
char *pass3 = password3;
char *pass4 = password4;

 while(*a ==*pass1){
  if(*a == '\0')
 {
 printf("password:%s\n", password1);
 break;
 }
  a++;
  pass1++;
  }
 while(*b ==*pass2){
  if(*b == '\0')
 {
 printf("password:%s\n", password2);
 break;
  }
 b++;
 pass2++;
 }
 while(*c ==*pass3){
  if(*c == '\0')
{
printf("password:%s\n", password3);
break;
}
c++;
pass3++;
}
while(*d ==*pass4){
  if(*d == '\0')
{
printf("password: %s\n",password4);
return 1;
}
d++;
pass4++;
}
return 0;
}

__global__ void kernel(){
char i1, i2;

char password[7];
password[6] ='\0';

int i = blockIdx.x +65;
int j = threadIdx.x+65;
char firstMatch =i;
char secondMatch =j;

password[0] =firstMatch;
password[1] =secondMatch;
for(i1='0'; i1<='9'; i1++){
for(i2='0'; i2<='9'; i2++){

password[2] =i1;
password[3] =i2;

if(is_a_match(password)){
}
else{
//printf("tried: %s\n",password);
}
     }
    }
  }


int time_difference(struct timespec *start, struct timespec *finish,long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

 struct timespec start, finish;  
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel<<<26,26>>>();
hipDeviceSynchronize();
 

 clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
         (time_elapsed/1.0e9));
return 0;
}
